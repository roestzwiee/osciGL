#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/* sounID project which demonstrates the basics on how to setup a project
* example application.
* Host code.
*/
#include ""
#include "hip/device_functions.h"


// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples

#include <mmsystem.h>
#pragma comment(lib, "winmm.lib")

void poo()
{

    // const int NUMPTS = 44100 * 10;              // 10 seconds
    // int sampleRate = 44100;
    // short int waveIn[NUMPTS];                   // 'short int' is a 16-bit type; I request 16-bit samples below
				// 								// for 8-bit capture, you'd use 'unsigned char' or 'BYTE' 8-bit types
    //
    // HWAVEIN     hWaveIn;
    // WAVEHDR     WaveInHdr;
    // MMRESULT    result;
    //
    // // Specify recording parameters
    // WAVEFORMATEX pFormat;
    // pFormat.wFormatTag = WAVE_FORMAT_PCM;       // simple, uncompressed format
    // pFormat.nChannels = 1;                      //  1=mono, 2=stereo
    // pFormat.nSamplesPerSec = sampleRate;        // 44100
    // pFormat.nAvgBytesPerSec = sampleRate * 2;   // = nSamplesPerSec * n.Channels * wBitsPerSample/8
    // pFormat.nBlockAlign = 2;                    // = n.Channels * wBitsPerSample/8
    // pFormat.wBitsPerSample = 16;                //  16 for high quality, 8 for telephone-grade
    // pFormat.cbSize = 0;
    //
    // result = waveInOpen(&hWaveIn, WAVE_MAPPER, &pFormat,
    //     0L, 0L, WAVE_FORMAT_DIRECT);
    // if (result)
    // {
    //     char fault[256];
    //     waveInGetErrorText(result, fault, 256);
    //     printf("Failed to open waveform input device.");
    //     return;
    // }
    //
    // // Set up and prepare header for input
    // WaveInHdr.lpData = (LPSTR)waveIn;
    // WaveInHdr.dwBufferLength = NUMPTS * 2;
    // WaveInHdr.dwBytesRecorded = 0;
    // WaveInHdr.dwUser = 0L;
    // WaveInHdr.dwFlags = 0L;
    // WaveInHdr.dwLoops = 0L;
    // waveInPrepareHeader(hWaveIn, &WaveInHdr, sizeof(WAVEHDR));
    //
    // // Insert a wave input buffer
    // result = waveInAddBuffer(hWaveIn, &WaveInHdr, sizeof(WAVEHDR));
    // if (result)
    // {
    //     printf("Failed to read block from device");
    //     return;
    // }
    //
    //
    // // Commence sampling input
    // result = waveInStart(hWaveIn);
    // if (result)
    // {
    //     printf("Failed to start recording");
    //     return;
    // }
    //
    //
    // // Wait until finished recording
    // do {
    //
    //     printf("Recording...\n");
    //
    // } while (waveInUnprepareHeader(hWaveIn, &WaveInHdr, sizeof(WAVEHDR)) == WAVERR_STILLPLAYING);
    //
    // waveInClose(hWaveIn);
    //
    // printf("Recording finished");

}


///////////////////////////////////////////////////////////////////////////////
//! Simple kernel to modify vertex positions in sine wave pattern
//! @param data  data in global memory
///////////////////////////////////////////////////////////////////////////////
__global__ void simple_vbo_kernel(float4* pos, unsigned int width, unsigned int height, float time)
{
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;

    // calculate uv coordinates
    float u = x / (float)width;
    float v = y / (float)height;
    u = u * 2.0f - 1.0f;
    v = v * 2.0f - 1.0f;

    // calculate simple sine wave pattern
    float freq = 4.0f;
    float w = sinf(u * freq + time) * cosf(v * freq + time) * 0.5f;

    // write output vertex
    pos[y * width + x] = make_float4(u, w, v, 1.0f);
}

extern "C"
void launch_kernel(float4 * pos, unsigned int mesh_width,
    unsigned int mesh_height, float time)
{
    // execute the kernel
    dim3 block(8, 8, 1);
    dim3 grid(mesh_width / block.x, mesh_height / block.y, 1);
    simple_vbo_kernel << < grid, block >> > (pos, mesh_width, mesh_height, time);
}