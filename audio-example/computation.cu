#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////
//
// Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
//
// Please refer to the NVIDIA end user license agreement (EULA) associated
// with this source code for terms and conditions that govern your use of
// this software. Any use, reproduction, disclosure, or distribution of
// this software and related documentation outside the terms of the EULA
// is strictly prohibited.
//
////////////////////////////////////////////////////////////////////////////

/* sounID project which demonstrates the basics on how to setup a project
* example application.
* Host code.
*/
#include ""
#include "hip/device_functions.h"


// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes CUDA
#include <hip/hip_runtime.h>

// includes, project
#include <hip/hip_runtime_api.h>
#include <helper_functions.h> // helper functions for SDK examples


///////////////////////////////////////////////////////////////////////////////
//! Simple kernel to modify vertex positions in sine wave pattern
//! @param data  data in global memory
///////////////////////////////////////////////////////////////////////////////
__global__ void simple_vbo_kernel_convert(short* input, float4* pos, unsigned int width, unsigned int height, float time)
{	
    unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
    
    // calculate uv coordinates
    float u = x / (float)width;
    float v;
    u = u * 2.0f - 1.0f;
    v = input[y * width + x];// / (float)2; // this call lets the kernel crash!
    
    // write output vertex
    pos[y * width + x] = make_float4(u, v / SHRT_MAX, 0.0f, 1.0f);
}

extern "C"
void launch_kernel_convert(short* input, float4 * pos, unsigned int mesh_width,
    unsigned int mesh_height, float time)
{
	// TODO: find a better way to limit the framerate...
    //Sleep(5);
	
    dim3 block(8, 1, 1);

    const int gridSizeX = mesh_width / block.x;
    const int gridSizeY = mesh_height / block.y;

	// TODO: Add further validation!
	if(gridSizeX == 0 || gridSizeY == 0)
	{
        printf("Error on grid size!\n");
        return;
	}
	
    dim3 grid(gridSizeX, gridSizeY, 1);


    checkCudaErrors(hipGetLastError());
	
    simple_vbo_kernel_convert <<< grid, block >>> (input, pos, mesh_width, mesh_height, time);

    checkCudaErrors(hipPeekAtLastError());
	
}